#include "hip/hip_runtime.h"
#include "include/redcufhe_gpu.cuh"
#include "include/details/error_gpu.cuh"

using namespace redcufhe;

#include <omp.h>
#include <stdlib.h>
#include <time.h>
#include <utility>
#include <vector>
#include <math.h>
#include <iostream>
#include <ctime>
#include <ratio>
#include <chrono>
#include <fstream>
#include <sstream>

using namespace std;
using namespace std::chrono;

PriKey pri_key;
uint32_t kNumTests;
PubKey bk;

// plaintext modulus
uint32_t message_space = 4096;

// shared vector used to issue/receive commands
vector<vector<pair<int, int>>> requests;


void setup(uint32_t kNumSMs, Ctxt** inputs, int32_t** pt, Stream** st, int idx) {
  hipSetDevice(idx);

  // send bootstrapping key to GPU
  Initialize(bk);

  // create CUDA streams for the GPU
  st[idx] = new Stream[kNumSMs];
  for (int i = 0; i < kNumSMs; i++) {
    st[idx][i].Create();
  }
  Synchronize();

  // Allocate memory for ciphertexts and encrypt
  (*inputs) = new Ctxt[2 * kNumTests];
  
  
  for (int i = 0; i < 2 * kNumTests; i++) {
    EncryptIntRed((*inputs)[i], pt[idx][i], message_space, pri_key);
  }
  Synchronize();
  return;
}

void server(int shares, uint32_t kNumSMs, int idx, Ctxt** answers, Stream** st) {
  while(1) {
    for (int i = 0; i < shares; i++) {
      // check for assignment
      if (requests[idx][i].first != -1) {
        // terminate upon kill signal (-2)
        if (requests[idx][i].first == -2) {
          Synchronize();
          return;
        }
        // Perform leveled addition
        AddRed((*answers)[requests[idx][i].second], (*answers)[requests[idx][i].second], (*answers)[requests[idx][i].first], st[idx][i % kNumSMs]);
        // clear assignment
        requests[idx][i].first = -1;
        requests[idx][i].second = -1;
      }
    }
  }
}

void AddCheck(int32_t& out, const int32_t& in0, const int32_t& in1) {
    //cout << "AddCheck: " << in0 << " + " << in1;
    out = in0 + in1;
    //cout << " = " << out <<  endl;
}


// GPU streams and memory management
__global__ void InitializeGpus(PriKey* sk_d, PubKey* pk_d, PriKey sk, PubKey pk) {
    *sk_d = sk;
    *pk_d = pk;
}

void ReadDataToArrays(const string& filename, vector<int>& array1, vector<int>& array2){
	ifstream infile(filename);
	string line;
	while (getline(infile, line)){
		stringstream ss(line);
		string value;
		int num1, num2;
		
		if (getline(ss, value, ',')){
		num1 = stoi(value);
		}
		
		if (getline(ss, value, ',')){
		num2 = stoi(value);
		}
		
		array1.push_back(num1);
		array2.push_back(num2);
	
	}
	
	infile.close();
}




int main() {
    
    srand(time(NULL));

   // get GPU stats (WARNING: assumes all GPUs have the same number of SMs)
    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t kNumSMs = prop.multiProcessorCount;
    
    
   // get number of available GPUs
    int numGPUs = 0;
    hipGetDeviceCount(&numGPUs);
    std::cout << "Available GPUs = " << numGPUs << endl;

    
    // create 2D array of plaintext and streams
    int32_t* pt[numGPUs]; // int is used instead of binary Ptxt
    Stream* st[numGPUs];
  
    // generate keyset
    SetSeed();
    PriKeyGen(pri_key);
    PubKeyGen(bk, pri_key);
    
    // getting secret numbers
    string filename = "test_data.txt";
    vector<int> array1, array2;
    
    ReadDataToArrays(filename, array1, array2);
    
    kNumTests = array1.size();
    for (int i = 0; i < numGPUs; i++) {
    	pt[i] = new int32_t[2 * kNumTests];
    	for (int j = 0; j < kNumTests; j++){
    		pt[i][j] = array1[j];
    	}
    	for (int j = 0; j < kNumTests; j++){
    		pt[i][j + kNumTests] = array2[j];
    	}	
     }
    
    // Initialize shared vector for thread communication
    int num_threads = numGPUs;
    requests.resize(num_threads);
    for (int i = 0; i < num_threads; i++) {
    	requests[i].resize(kNumTests);
    	for (int j = 0; j < kNumTests; j++) {
      // each element holds indices of data array
      requests[i][j] = make_pair(-1,-1);
        }
     }
  
    Ctxt* answers[numGPUs];
    omp_set_num_threads(numGPUs);
    
    // timer t0
    high_resolution_clock::time_point t0 = high_resolution_clock::now();
    
    // Initialize data on each available GPU
    #pragma omp parallel for shared(st, answers)
    for (int i = 0; i < numGPUs; i++) {
    	setup(kNumSMs, &answers[i], pt, st, i);
     }

    // one worker thread for each GPU and a scheduler thread
    omp_set_num_threads(numGPUs+1);
    
    //timer t1 for set up
    high_resolution_clock::time_point t1 = high_resolution_clock::now();
    duration<double> time_span_s = duration_cast<duration<double>>(t1 - t0);
    duration<double, milli> time_span_ms = duration_cast<duration<double>>(t1 - t0);
    std::cout << "Time to set up: " << time_span_s.count() << " seconds" << endl;
    std::cout << "Time to set up: " << time_span_ms.count() << " milliseconds" << endl;


  /////////////////////////////////////////
  //
  // (RED)cuFHE Dynamic Scheduler
  // Enables automatic allocation of FHE
  // workloads to multiple GPUs
  //
  /////////////////////////////////////////
    #pragma omp parallel for shared(answers, st, requests)
    for (int i = 0; i < (num_threads+1); i++) {
    if (i != 0) { // workers
    	int thread_id = omp_get_thread_num() - 1;
      hipSetDevice(thread_id);
      server(kNumTests, kNumSMs, thread_id, &answers[i-1], st);
      Synchronize();
     }
    else { // master thread
      int turn = 1; // indicates target worker
      for (int j = 0; j < (kNumTests*numGPUs); j++) {
        if ((j % kNumTests == 0) && (j > 0)) {
          turn++; // assign to next worker
          if (turn > num_threads) { // excludes scheduler
            turn = 1;
          }
        }
        // assign input 1 as index j of GPU array
        requests[turn-1][j % kNumTests].second = j % (kNumTests);
        // assign input 2 as index j+kNumTests
        requests[turn-1][j % kNumTests].first = ((j%kNumTests)+kNumTests) % (2*kNumTests);
      }
      // check to see if all threads are done
      bool end = false;
      while (end == false) {
        end = true;
        for (int j = 0; j < num_threads; j++) {
          for (int k = 0; k < kNumTests; k++) {
            if (requests[j][k].first != -1) {
              end = false;
              break;
            	}
             }
           }
        }
      // terminate workers
      for (int j = 0; j < num_threads; j++) {
        for (int k = 0; k < kNumTests; k++) {
          requests[j][k].first = -2;
          }
        }
     }
     }
  	high_resolution_clock::time_point t2 = high_resolution_clock::now();
  
      std::cout << "Arithmetic evals: " << kNumTests*numGPUs << endl;

  // Confirm results and check for errors
  int wrong_counter[numGPUs];
  omp_set_num_threads(numGPUs);
  
  high_resolution_clock::time_point t3 , t4 ;
  #pragma omp parallel shared(wrong_counter)
  {
    int32_t* recovered_pt = new int32_t[kNumTests];
    int thread_num = omp_get_thread_num();
    hipSetDevice(thread_num);
    
    
    for (int i = 0; i < kNumTests; i++) {
      AddCheck(pt[thread_num][i], pt[thread_num][i+kNumTests], pt[thread_num][i]);
    }
    
    t3 = high_resolution_clock::now();
    
    for (int i = 0; i < kNumTests; i++) {
    DecryptIntRed(recovered_pt[i], answers[thread_num][i+kNumTests], message_space, pri_key);
    }
    t4 = high_resolution_clock::now();
    
    wrong_counter[thread_num] = 0;
    for (int i = 0; i < kNumTests; i++) {
      if (pt[thread_num][i+kNumTests] != recovered_pt[i]) {
        cout << "Expected: " << pt[thread_num][i+kNumTests] << "  Actual: " << recovered_pt[i] << endl;
        wrong_counter[thread_num]++;
      }
    }
    delete [] recovered_pt;
  }
  
  
  //timer t2 for arithmetic 
  duration<double> time_span2_s = duration_cast<duration<double>>(t2 - t1);
  duration<double, milli> time_span2_ms = duration_cast<duration<double>>(t2 - t1);
  std::cout << "Time to atirhmetic: " << time_span2_s.count() << " seconds" << endl;
  std::cout << "Time to atirhmetic: " << time_span2_ms.count() << " milliseconds" << endl;
  
  duration<double> time_span3_s = duration_cast<duration<double>>(t4 - t3);
  duration<double, milli> time_span3_ms = duration_cast<duration<double>>(t4 - t3);
  std::cout << "Time to decrypt: " << time_span3_s.count() << " seconds" << endl;
  std::cout << "Time to decrypt: " << time_span3_ms.count() << " milliseconds" << endl;

  std::cout << "kNumTests :" << kNumTests << " , num_threads : " << num_threads + 1 << endl;
  
  

  for (int i = 0; i < numGPUs; i++) {
    cout << "GPU #" << i << " errors: " << wrong_counter[i] << endl;
  }

  

  for (int i = 0; i < numGPUs; i++) {
    delete [] pt[i];
  }
  // free GPU memory
  CleanUp();
  std::cout << endl ;
  if ( wrong_counter[0] != 0 )
  	return 1;
  
  return 0;
}
